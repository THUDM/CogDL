#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>
#include <vector>

#define MAX(a, b) ((a < b) ? (b) : (a))

__global__ void edgeconv_forward_kernel(
    const int m, const int f,
    const int k, const int *src_ind,
    const float *h_src, const float *h_dst,
    float *out_feat,
    float *max_idx)
{
    int rid = blockIdx.x;
    int lb = rid * k;
    int hb = (rid + 1) * k;
    int fid = threadIdx.y;
    int ptr = lb + threadIdx.x;
    int loop = (k+31) / 32;

    for (; fid < f; fid += 32)
    {
        float max_val = -1e39;
        float max_id = -1;
        float dst_val = h_dst[rid * f + fid];
        for (int j = 0; j < loop; j++)
        {
            int pid = ptr + (j << 5);
            float feat = -1e38;
            int src_id = -1;
            if (pid < hb)
            {
                src_id = src_ind[pid];
                feat = dst_val + h_src[src_id * f + fid];
            }
            __syncwarp();
            for (int stride = 16; stride > 0; stride >>= 1)
            {
                float tmp = __shfl_xor_sync(0xffffffff, feat, stride, 32);
                float tmp_id = __shfl_xor_sync(0xffffffff, src_id, stride, 32);
                if (tmp >= feat)
                {
                    feat = tmp;
                    src_id = tmp_id;
                }
            }
            if (feat >= max_val)
            {
                max_val = feat;
                max_id = src_id;
            }
        }
        if (threadIdx.x == 0)
        {
            out_feat[rid * f + fid] = max_val;
            max_idx[rid * f + fid] = max_id;
        }
    }
}

std::vector<torch::Tensor> edgeconv_forward_cuda(
    const int k,
    const torch::Tensor src_ind,
    const torch::Tensor h_src,
    const torch::Tensor h_dst)
{
    const auto m = h_src.size(0);
    const auto f = h_src.size(1);
    auto devid = src_ind.device().index();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
    auto out_feat = torch::empty({m, f}, options);
    auto max_idx = torch::empty({m, f}, options);
    dim3 grids(m, 1, 1);
    dim3 blocks;
    if (f < 32)
    {
        blocks = dim3(32, f, 1);
    }
    else
    {
        blocks = dim3(32, 32, 1);
    }
    edgeconv_forward_kernel<<<grids, blocks>>>(m, f, k, src_ind.data_ptr<int>(), h_src.data_ptr<float>(), h_dst.data_ptr<float>(), out_feat.data_ptr<float>(), max_idx.data_ptr<float>());
    return {out_feat, max_idx};
}

__global__ void edgeconv_backward_kernel(
    const int m, const int f,
    const int *max_idx,
    const float *grad_out,
    float *grad_src)
{
    int src_id = blockIdx.x;
    int fid = threadIdx.x;
    int ptr = src_id * f + fid;
    grad_src[ptr] = 0;
    atomicAdd(&grad_src[max_idx[ptr] * f + fid], grad_out[ptr]);
    // grad_src[max_idx[ptr] * f + fid] += grad_out[ptr];
}

torch::Tensor edgeconv_backward_cuda(
    const torch::Tensor grad_out,
    const torch::Tensor max_idx)
{
    auto m = grad_out.size(0);
    auto f = grad_out.size(1);
    auto devid = grad_out.device().index();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
    auto grad_src = torch::empty({m, f}, options);
    edgeconv_backward_kernel<<<dim3(m, 1, 1), dim3(f, 1, 1)>>>(m, f, max_idx.data_ptr<int>(), grad_out.data_ptr<float>(), grad_src.data_ptr<float>());
    return grad_src;
}

__global__ void edgeconv_forward_kernel_110(
    const int m, const int f,
    const int k, const int *src_ind,
    const float *h_src, const float *h_dst,
    float* edge_feat,
    float *out_feat,
    float *max_idx)
{
    int rid = blockIdx.x;
    int lb = rid * k;
    int hb = (rid + 1) * k;
    int fid = threadIdx.y;
    int ptr = lb + threadIdx.x;
    int loop = k / 32;

    for (; fid < f; fid += 32)
    {
        float max_val = -1e39;
        float max_id = -1;
        float dst_val = h_dst[rid * f + fid];
        for (int j = 0; j < loop; j++)
        {
            int pid = ptr + (j << 5);
            float feat = -1e38;
            int src_id = -1;
            if (pid < hb)
            {
                src_id = src_ind[pid];
                feat = dst_val + h_src[src_id * f + fid];
                edge_feat[pid*f+fid]=feat;
            }
            __syncwarp();
            for (int stride = 16; stride > 0; stride >>= 1)
            {
                float tmp = __shfl_xor_sync(0xffffffff, feat, stride, 32);
                float tmp_id = __shfl_xor_sync(0xffffffff, src_id, stride, 32);
                if (tmp >= feat)
                {
                    feat = tmp;
                    src_id = tmp_id;
                }
            }
            if (feat >= max_val)
            {
                max_val = feat;
                max_id = src_id;
            }
        }
        if (threadIdx.x == 0)
        {
            out_feat[rid * f + fid] = max_val;
            max_idx[rid * f + fid] = max_id;
        }
    }
}

std::vector<torch::Tensor> edgeconv_forward_cuda_110(
    const int k,
    const torch::Tensor src_ind,
    const torch::Tensor h_src,
    const torch::Tensor h_dst)
{
    const auto m = h_src.size(0);
    const auto f = h_src.size(1);
    auto devid = src_ind.device().index();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
    auto out_feat = torch::empty({m, f}, options);
    auto max_idx = torch::empty({m, f}, options);
    dim3 grids(m, 1, 1);
    dim3 blocks;
    if (f < 32)
    {
        blocks = dim3(32, f, 1);
    }
    else
    {
        blocks = dim3(32, 32, 1);
    }
    edgeconv_forward_kernel<<<grids, blocks>>>(m, f, k, src_ind.data_ptr<int>(), h_src.data_ptr<float>(), h_dst.data_ptr<float>(), out_feat.data_ptr<float>(), max_idx.data_ptr<float>());
    return {out_feat, max_idx};
}

__global__ void edgeconv_backward_kernel_110(
    const int m, const int f,
    const int *max_idx,
    const float *grad_out,
    float *grad_src)
{
    int src_id = blockIdx.x;
    int fid = threadIdx.x;
    int ptr = src_id * f + fid;
    grad_src[ptr] = 0;
    atomicAdd(&grad_src[max_idx[ptr] * f + fid], grad_out[ptr]);
    // grad_src[max_idx[ptr] * f + fid] += grad_out[ptr];
}

torch::Tensor edgeconv_backward_cuda_110(
    const torch::Tensor grad_out,
    const torch::Tensor max_idx)
{
    auto m = grad_out.size(0);
    auto f = grad_out.size(1);
    auto devid = grad_out.device().index();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
    auto grad_src = torch::empty({m, f}, options);
    edgeconv_backward_kernel<<<dim3(m, 1, 1), dim3(f, 1, 1)>>>(m, f, max_idx.data_ptr<int>(), grad_out.data_ptr<float>(), grad_src.data_ptr<float>());
    return grad_src;
}
